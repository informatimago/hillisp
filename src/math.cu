#include "hip/hip_runtime.h"
#include "lisp.h"

template<typename T>
__global__ void 
xd_add(const T* __restrict__ a, const T* __restrict__ b, T* __restrict__ c, const size_t size) {
  for (int i = TID; i < size; i += STRIDE)
    c[i] = a[i] + b[i];
}

template<typename T>
__global__ void
xd_sub(const T* __restrict__ a, const T* __restrict__ b, T* __restrict__ c, const size_t size) {
  for (int i = TID; i < size; i += STRIDE)
    c[i] = a[i] - b[i];
}

template<typename T>
__global__ void
xd_mul(const T* __restrict__ a, const T* __restrict__ b, T* __restrict__ c, const size_t size) {
  for (int i = TID; i < size; i += STRIDE)
    c[i] = a[i] * b[i];
}

template<typename T>
__global__ void
xd_div(const T* __restrict__ a, const T* __restrict__ b, T* __restrict__ c, const size_t size) {
  for (int i = TID; i < size; i += STRIDE)
    c[i] = a[i] / b[i];
}

template<typename T>
__global__ void
xd_fma(const T* __restrict__ a, const T* __restrict__ b, const T* __restrict__ c, T* __restrict__ d, const size_t size) {
  for (int i = TID; i < size; i += STRIDE)
    d[i] = a[i] * b[i] + c[i];
}

x_any _x_add(x_any a, x_any b, bool assign) {
  x_any c;
  if (are_ixectors(a, b)) {
    assert_xectors_align(a, b);
    if (assign)
      c = a;
    else
      c = new_ixector(xector_size(a));

    SYNCS(x_env.stream);
    xd_add<int64_t><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<int64_t>(a), cars<int64_t>(b), cars<int64_t>(c), xector_size(a));
    CHECK;
    return c;
  }
  else if (are_dxectors(a, b)) {
    assert_xectors_align(a, b);
    if (assign)
      c = a;
    else
      c = new_dxector(xector_size(a));
    SYNCS(x_env.stream);
    xd_add<double><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<double>(a), cars<double>(b), cars<double>(c), xector_size(a));
    CHECK;
    return c;
  }
  else if (are_ints(a, b)) {
    if (assign) {
      set_val(a, ival(a) + ival(b));
      return a;
    }
    else {
      return new_int(ival(a) + ival(b));
    }
  }
  else if (are_doubles(a, b)) {
    if (assign) {
      dval(a) = dval(a) + dval(b);
      return a;
    }
    else {
      return new_double(dval(a) + dval(b));
    }
  }
  else if (are_dcomplex(a, b)) {
    if (assign) {
      cval(a) = hipCadd(cval(a), cval(b));
      return a;
    }
    else {
      return new_dcomplex(hipCadd(cval(a), cval(b)));
    }
  }
  assert(0);
  return x_env.nil;
}

x_any x_add(x_any a, x_any b) {
  return _x_add(a, b, false);
}

x_any x_addass(x_any a, x_any b) {
  return _x_add(a, b, true);
}

x_any _x_sub(x_any a, x_any b, bool assign) {
  x_any c;
  if (are_ixectors(a, b)) {
    assert_xectors_align(a, b);
    if (assign)
      c = a;
    else
      c = new_ixector(xector_size(a));

    SYNCS(x_env.stream);
    xd_sub<int64_t><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<int64_t>(a), cars<int64_t>(b), cars<int64_t>(c), xector_size(a));
    CHECK;
    return c;
  }
  else if (are_dxectors(a, b)) {
    assert_xectors_align(a, b);
    if (assign)
      c = a;
    else
      c = new_dxector(xector_size(a));
    SYNCS(x_env.stream);
    xd_sub<double><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<double>(a), cars<double>(b), cars<double>(c), xector_size(a));
    CHECK;
    return c;
  }
  else if (are_ints(a, b)) {
    if (assign) {
      set_val(a, ival(a) - ival(b));
      return a;
    }
    else {
      return new_int(ival(a) - ival(b));
    }
  }
  else if (are_doubles(a, b)) {
    if (assign) {
      dval(a) = dval(a) - dval(b);
      return a;
    }
    else {
      return new_double(dval(a) - dval(b));
    }
  }
  else if (are_dcomplex(a, b)) {
    if (assign) {
      cval(a) = hipCsub(cval(a), cval(b));
      return a;
    }
    else {
      return new_dcomplex(hipCsub(cval(a), cval(b)));
    }
  }
  assert(0);
  return x_env.nil;
}


x_any x_sub(x_any a, x_any b) {
  return _x_sub(a, b, false);
}

x_any x_subass(x_any a, x_any b) {
  return _x_sub(a, b, true);
}

x_any _x_mul(x_any a, x_any b, bool assign) {
  x_any c;
  if (are_ixectors(a, b)) {
    assert_xectors_align(a, b);
    if (assign)
      c = a;
    else
      c = new_ixector(xector_size(a));
    SYNCS(x_env.stream);
    xd_mul<int64_t><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<int64_t>(a), cars<int64_t>(b), cars<int64_t>(c), xector_size(a));
    CHECK;
    return c;
  }
  else if (are_dxectors(a, b)) {
    assert_xectors_align(a, b);
    if (assign)
      c = a;
    else
      c = new_dxector(xector_size(a));
    SYNCS(x_env.stream);
    xd_mul<double><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<double>(a), cars<double>(b), cars<double>(c), xector_size(a));
    CHECK;
    return c;
  }
  else if (are_ints(a, b)) {
    if (assign) {
      set_val(a, ival(a) * ival(b));
      return a;
    } else {
      return new_int(ival(a) * ival(b));
    }
  }
  else if (are_doubles(a, b)) {
    if (assign) {
      dval(a) = dval(a) * dval(b);
      return a;
    } else {
      return new_double(dval(a) * dval(b));
    }
  }
  else if (are_dcomplex(a, b)) {
    if (assign) {
      cval(a) = hipCmul(cval(a), cval(b));
    } else {
      return new_dcomplex(hipCmul(cval(a), cval(b)));
    }
  }
  assert(0);
  return x_env.nil;
}

x_any x_mul(x_any a, x_any b) {
  return _x_mul(a, b, false);
}

x_any x_mulass(x_any a, x_any b) {
  return _x_mul(a, b, true);
}

x_any _x_div(x_any a, x_any b, bool assign) {
  x_any c;
  if (are_ixectors(a, b)) {
    assert_xectors_align(a, b);
    if (assign)
      c = a;
    else
      c = new_ixector(xector_size(a));
    SYNCS(x_env.stream);
    xd_div<int64_t><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<int64_t>(a), cars<int64_t>(b), cars<int64_t>(c), xector_size(a));
    CHECK;
    return c;
  }
  else if (are_dxectors(a, b)) {
    assert_xectors_align(a, b);
    if (assign)
      c = a;
    else
      c = new_dxector(xector_size(a));
    SYNCS(x_env.stream);
    xd_div<double><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<double>(a), cars<double>(b), cars<double>(c), xector_size(a));
    CHECK;
    return c;
  }
  else if (are_ints(a, b)) {
    if (assign) {
      set_val(a, ival(a) / ival(b));
      return a;
    } else {
      return new_int(ival(a) / ival(b));
    }
  }
  else if (are_doubles(a, b)) {
    if (assign) {
      dval(a) = dval(a) / dval(b);
      return a;
    } else {
      return new_double(dval(a) / dval(b));
    }
  }
  else if (are_dcomplex(a, b)) {
    if (assign) {
      cval(a) = hipCdiv(cval(a), cval(b));
      return a;
    } else {
      return new_dcomplex(hipCdiv(cval(a), cval(b)));
    }
  }
  assert(0);
  return x_env.nil;
}

x_any x_div(x_any a, x_any b) {
  return _x_div(a, b, false);
}

x_any x_divass(x_any a, x_any b) {
  return _x_div(a, b, true);
}

x_any _x_fma(x_any a, x_any b, x_any c, bool assign) {
  x_any d;
  if (are_ixectors(a, b)) {
    assert_xectors_align(a, b);
    assert_xectors_align(a, c);
    if (assign)
      d = a;
    else
      d = new_ixector(xector_size(a));
    SYNCS(x_env.stream);
    xd_fma<int64_t><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<int64_t>(a), cars<int64_t>(b), cars<int64_t>(c), cars<int64_t>(d), xector_size(a));
    CHECK;
    return d;
  }
  else if (are_ints(a, b)) {
    if (assign) {
      set_val(a, ival(a) * ival(b) + ival(c));
      return a;
    } else {
      return new_int(ival(a) * ival(b) + ival(c));
    }
  }
  else if (are_doubles(a, b)) {
    if (assign) {
      dval(a) = dval(a) * dval(b) + dval(c);
      return a;
    } else {
      return new_double(dval(a) * dval(b) + dval(c));
    }
  }
  else if (are_dcomplex(a, b)) {
    if (assign) {
      cval(a) = hipCadd(hipCmul(cval(a), cval(b)), cval(c));
      return a;
    } else {
      return new_dcomplex(hipCadd(hipCmul(cval(a), cval(b)), cval(c)));
    }
  }
  assert(0);
  return x_env.nil;
}

x_any x_fma(x_any a, x_any b, x_any c) {
  return _x_fma(a, b, c, false);
}

x_any x_fmaass(x_any a, x_any b, x_any c) {
  return _x_fma(a, b, c, true);
}

x_any x_complex(x_any real, x_any imag) {
  return new_dcomplex(make_hipDoubleComplex(dval(real), dval(imag)));
}
