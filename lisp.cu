#include "lisp.h"

char* new_name(const char* name) {
  char *n;
  hipMallocManaged(&n, strlen(name) + 1);
  assert(n != NULL);
  strcpy(n, name);
  return n;
}

x_any new_cell(const char* name) {
  x_any cell;
  hipMallocManaged(&cell, sizeof(x_cell));
  assert(cell != NULL);
  name(cell) = new_name(name);
  return cell;
}

x_any def_token(const char* new_name) {
  x_any cell;
  cell = new_cell(new_name);
  flags(cell) = TOKEN;
  return cell;
}

int hash(const char *name) {
  int value = 0;
  while (*name != '\0')
    value = (value * HASH_MULTIPLIER + *name++) % HASH_TABLE_SIZE;
  return value;
}

x_any lookup(const char *name, x_any cell) {
  if (cell == x_nil)
    return NULL;
  else if (strcmp(name(car(cell)), name) == 0)
    return car(cell);
  else
    return lookup(name, cdr(cell));
}

x_any create_symbol(const char *new_name) {
  x_any cell;
  cell = new_cell(new_name);
  flags(cell) = SYMBOL;
  return cell;
}

void print_cell(x_any cell, FILE *outfile) {
  if (is_atom(cell))
    fprintf(outfile, "%s", name(cell));
  else {
    putc('(', outfile);
    print_list(cell, outfile);
  }
}

void print_list(x_any cell, FILE *outfile) {
  print_cell(car(cell), outfile);
  if (cdr(cell) == x_nil)
    putc(')', outfile);
  else if (!is_pair(cdr(cell)) ) {
    fprintf(outfile, " . ");
    print_cell(cdr(cell), outfile);
    putc(')', outfile);
  }
  else {
    putc(' ', outfile);
    print_list(cdr(cell), outfile);
  }
}


void enter(x_any cell) {
  int hash_val;

  hash_val = hash(name(cell));
  hash_table[hash_val] = x_cons(cell, hash_table[hash_val]);
}

x_any intern(const char *name) {
  x_any cell;

  cell = lookup(name, hash_table[hash(name)]);
  if (cell != NULL)
    return cell;
  else {
    cell = create_symbol(name);
    enter(cell);
    return cell;
  }
}

int length(x_any cell) {
  if (cell == x_nil)
    return 0;
  else
    return 1 + length(cdr(cell));
}

x_any list_eval(x_any cell) {
  if (cell == x_nil)
    return x_nil;
  else
    return x_cons(x_eval(car(cell)), list_eval(cdr(cell)));
}

x_any def_builtin(char const *name, void *fn, size_t num_args) {
  x_any cell;

  cell = intern(name);
  flags(cell) = BUILTIN;
  data(cell) = fn;
  size(cell) = num_args;
  return cell;
}

x_any read_token(FILE *infile) {
  int c;
  static char buf[MAX_NAME_LEN];
  char *ptr = buf;

  do {
    c = getc(infile);
    if (c == ';')
      do c = getc(infile); while (c != '\n' && c != EOF);
  } while (isspace(c));
  switch (c) {
  case EOF:
    return x_eof;
  case '(':
    return x_left;
  case ')':
    return x_right;
  case '.':
    return x_dot;
  default:
    *ptr++ = c;
    while ((c = getc(infile)) != EOF && !isspace(c) && c != '(' && c != ')')
      *ptr++ = c;
    if (c != EOF)
      ungetc(c, infile);
    *ptr = '\0';
    return intern(buf);
  }
}

x_any read_cdr(FILE *infile) {
  x_any cdr;
  x_any token;

  cdr = read_sexpr(infile);
  token = read_token(infile);

  if (token == x_right)
    return cdr;
  else
    assert(0);
  return x_nil;
}

x_any read_tail(FILE *infile) {
  x_any token;
  x_any temp;

  token = read_token(infile);

  if (is_symbol(token) || is_builtin(token))
    return x_cons(token, read_tail(infile));

  if (token == x_left) {
    temp = read_head(infile);
    return x_cons(temp, read_tail(infile));
  }

  if (token == x_dot)
    return read_cdr(infile);

  if (token == x_right)
    return x_nil;

  if (token == x_eof)
    assert(0);
  return x_nil;
}

x_any read_head(FILE *infile) {
  x_any token;
  x_any temp;

  token = read_token(infile);
  if (is_symbol(token) || is_builtin(token))
    return x_cons(token, read_tail(infile));
  if (token == x_left) {
    temp = read_head(infile);
    return x_cons(temp, read_tail(infile));
  }
  if (token == x_right)
    return x_nil;
  if (token == x_dot)
    assert(0);
  if (token == x_eof)
    assert(0);
  return x_nil;
}

x_any read_sexpr(FILE *infile) {
  x_any token;

  token = read_token(infile);
  if (is_symbol(token) || is_builtin(token))
    return token;
  if (token == x_left)
    return read_head(infile);
  if (token == x_right)
    assert(0);
  if (token == x_dot)
    assert(0);
  if (token == x_eof)
    return token;
  return x_nil;
}

void init(void) {
  x_dot = def_token(".");
  x_left = def_token("(");
  x_right = def_token(")");
  x_eof = def_token("EOF");

  x_nil = create_symbol("nil");
  for (int i = 0; i < HASH_TABLE_SIZE; i++)
    hash_table[i] = x_nil;
  enter(x_nil);
  x_true = intern("true");

  def_builtin("is", (void*)x_is, 2);
  def_builtin("car", (void*)x_car, 1);
  def_builtin("cdr", (void*)x_cdr, 1);
  def_builtin("cons", (void*)x_cons, 2);
  def_builtin("quote", (void*)x_quote, 1);
  def_builtin("cond", (void*)x_cond, 1);
  def_builtin("eval", (void*)x_eval, 1);
  def_builtin("apply", (void*)x_apply, 2);
  def_builtin("print", (void*)x_print, 1);
}

int main(int argc, const char* argv[]) {
  x_any expr;
  x_any value;

  init();
  for (;;) {
    printf("? ");
    expr = read_sexpr(stdin);
    if (expr == x_eof)
      break;
    value = x_eval(expr);
    printf(": ");
    print_cell(value, stdout);
    putchar('\n');
  }
}
