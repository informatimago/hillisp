#include "hip/hip_runtime.h"
#include "lisp.h"

__thread x_environ x_env;

x_any c_alloc(x_any type) {
  x_any cell;
  if (!(cell = x_env.cell_pools->free))
    assert(0);
  x_env.cell_pools->free = car(cell);
  set_cdr(cell, NULL);
  set_car(cell, NULL);
  type(cell) = type;
  return cell;
}

void* x_alloc(size_t size) {
void* result;
  hipMallocManaged(&result, size);
  hipStreamAttachMemAsync(x_env.stream, result);
  CHECK;
  assert(result != NULL);
  return result;
}

char* new_name(const char* name) {
  char *n;
  n = (char*)malloc(strlen(name) + 1);
  strcpy(n, name);
  return n;
}

x_any new_cell(const char* name, x_any type) {
  x_any cell;
  cell = c_alloc(type);
  if (name == NULL)
    set_val(cell, NULL);
  else
    set_val(cell, new_name(name));
  return cell;
}

x_any new_int(int64_t value) {
  x_any cell;
  cell = new_cell(NULL, x_env.int_);
  set_val(cell, value);
  return cell;
}

x_cell_pool* new_cell_pool(x_cell_pool* old) {
  x_cell_pool* h;
  x_any cell;
  h = (x_cell_pool*)malloc(sizeof(x_cell_pool));
  h->next = old;
  cell = h->cells + X_YOUNG_CELL_POOL_SIZE - 1;
  do
    free_cell(h, cell);
  while (--cell >= h->cells);
  return h;
}

x_frame* new_frame() {
  x_frame * f;
  f = (x_frame*)malloc(sizeof(x_frame));
  f->next = NULL;
  f->prev = NULL;
  for (int i = 0; i < X_HASH_TABLE_SIZE; i++)
    f->names[i] = x_env.nil;
  return f;
}

int length(x_any cell) {
  if (cell == x_env.nil)
    return 0;
  else
    return 1 + length(cdr(cell));
}

x_any list_eval(x_any cell) {
  if (cell == x_env.nil)
    return x_env.nil;
  if (is_atom(cell))
    return cell;
  else
    return x_cons(x_eval(car(cell)), list_eval(cdr(cell)));
}

x_any read_token(FILE *infile) {
  int c;
  static char buf[X_MAX_NAME_LEN];
  char *ptr = buf;
  x_any cell;

  do {
    c = getc(infile);
    if (c == ';')
      do c = getc(infile); while (c != '\n' && c != EOF);
  } while (isspace(c));
  switch (c) {
  case EOF:
    return x_env.eof;
  case '(':
    return x_env.lparen;
  case ')':
    return x_env.rparen;
  case '[':
    return x_env.lbrack;
  case ']':
    return x_env.rbrack;
  case '.':
    return x_env.dot;
  default:
    *ptr++ = c;
    while ((c = getc(infile)) != EOF &&
           !isspace(c) &&
           c != '(' && c != ')' &&
           c != '[' && c!= ']')
      *ptr++ = c;
    if (c != EOF)
      ungetc(c, infile);
    *ptr = '\0';
    if (strcmp(buf, "symbol") == 0)
      return x_env.symbol;

    if (isdigit(buf[0]) || (buf[0] == '-' && isdigit(buf[1]))) {
      cell = new_int(atoll(buf));
      return cell;
    }
    return intern(buf);
  }
}

x_any read_cdr(FILE *infile) {
  x_any cdr;
  x_any token;
  cdr = read_sexpr(infile);
  token = read_token(infile);
  if (token == x_env.rparen)
    return cdr;
  else
    assert(0);
  return x_env.nil;
}

x_any read_sexpr_tail(FILE *infile) {
  x_any token;
  x_any temp;
  token = read_token(infile);
  if (is_atom(token))
    return x_cons(token, read_sexpr_tail(infile));
  if (token == x_env.lparen) {
    temp = read_sexpr_head(infile);
    return x_cons(temp, read_sexpr_tail(infile));
  }
  if (token == x_env.lbrack) {
    temp = read_xector(infile);
    return x_cons(temp, read_sexpr_tail(infile));
  }
  if (token == x_env.dot)
    return read_cdr(infile);
  if (token == x_env.rparen)
    return x_env.nil;
  if (token == x_env.eof)
    assert(0);
  return x_env.nil;
}

x_any read_sexpr_head(FILE *infile) {
  x_any token;
  x_any temp;
  token = read_token(infile);
  if (is_atom(token))
    return x_cons(token, read_sexpr_tail(infile));
  else if (token == x_env.lparen) {
    temp = read_sexpr_head(infile);
    return x_cons(temp, read_sexpr_tail(infile));
  }
  else if (token == x_env.lbrack) {
    temp = read_xector(infile);
    return x_cons(temp, read_sexpr_tail(infile));
  }
  else if (token == x_env.rparen)
    return x_env.nil;
  else if (token == x_env.dot)
    assert(0);
  else if (token == x_env.eof)
    assert(0);
  return x_env.nil;
}

x_any read_xector(FILE *infile) {
  x_any val;
  x_any cell;
  x_any typ = NULL;
  size_t size = 0;
  cell = new_xector<int64_t>(NULL, X_XECTOR_BLOCK_SIZE);
  do {
    val = x_eval(read_sexpr(infile));
    if (val == x_env.nil)
      break;
    if (typ == NULL)
      typ = type(val);
    else if (type(val) != typ)
      assert(0); // must all be same type

    if (typ == x_env.int_)
      xector_set_car_ith(cell, size, ival(val));
    else if (typ == x_env.xector)
      xector_set_car_ith(cell, size, car(val));
    else
      assert(0);
    size++;
  } while (1);
  car(cell) = new_int(size);
  return cell;
}

x_any read_sexpr(FILE *infile) {
  x_any token;
  token = read_token(infile);
  if (is_atom(token))
    return token;
  if (token == x_env.lbrack)
    return read_xector(infile);
  if (token == x_env.lparen)
    return read_sexpr_head(infile);
  if (token == x_env.rparen)
    assert(0);
  if (token == x_env.dot)
    assert(0);
  if (token == x_env.eof)
    return token;
  return x_env.nil;
}

x_any def_token(const char* new_name) {
  return new_cell(new_name, x_env.token);
}

x_any def_builtin(char const *name, void *fn, size_t num_args, void *dfn) {
  x_any cell;
  cell = intern(name);
  type(cell) = x_env.builtin;
  set_val(cell, fn);
  switch(num_args) {
  case 0:
    type(cell) = x_env.fn0;
    break;
  case 1:
    type(cell) = x_env.fn1;
    break;
  case 2:
    type(cell) = x_env.fn2;
    break;
  case 3:
    type(cell) = x_env.fn3;
    break;
  }
  return cell;
}

void init(void) {
  x_env.cell_pools = new_cell_pool(NULL);

  x_env.symbol = new_cell("symbol", NULL);
  type(x_env.symbol) = x_env.symbol;
  x_env.pair = new_cell("pair", NULL);
  x_env.nil = new_cell("nil", x_env.symbol);

  x_env.frames = new_frame();

  bind("nil", x_env.nil, x_env.frames);
  bind("symbol", x_env.symbol, x_env.frames);
  bind("pair", x_env.pair, x_env.frames);

  x_env.binding = intern("binding");
  x_env.token = intern("token");
  x_env.builtin = intern("builtin");
  x_env.user = intern("user");
  x_env.true_ = intern("true");
  x_env.xector = intern("xector");
  x_env.int_ = intern("int");
  x_env.str = intern("str");

  x_env.fn0 = intern("fn0");
  x_env.fn1 = intern("fn1");
  x_env.fn2 = intern("fn2");
  x_env.fn3 = intern("fn3");

  x_env.dot = def_token(".");
  x_env.lparen = def_token("(");
  x_env.rparen = def_token(")");
  x_env.lbrack = def_token("[");
  x_env.rbrack = def_token("]");
  x_env.eof = def_token("EOF");

  def_builtin("is", (void*)x_is, 2, NULL);
  def_builtin("isinstance", (void*)x_isinstance, 2, NULL);
  def_builtin("type", (void*)x_type, 1, NULL);
  def_builtin("car", (void*)x_car, 1, NULL);
  def_builtin("cdr", (void*)x_cdr, 1, NULL);
  def_builtin("cons", (void*)x_cons, 2, NULL);
  def_builtin("quote", (void*)x_quote, 1, NULL);
  def_builtin("if", (void*)x_if, 1, NULL);
  def_builtin("while", (void*)x_while, 1, NULL);
  def_builtin("eval", (void*)x_eval, 1, NULL);
  def_builtin("apply", (void*)x_apply, 2, NULL);
  def_builtin("assert", (void*)x_assert, 1, NULL);
  def_builtin("print", (void*)x_print, 1, NULL);
  def_builtin("println", (void*)x_println, 1, NULL);
  def_builtin("+", (void*)x_add, 2, NULL);
  def_builtin("-", (void*)x_sub, 2, NULL);
  def_builtin("*", (void*)x_mul, 2, NULL);
  def_builtin("/", (void*)x_div, 2, NULL);
  def_builtin("fma", (void*)x_fma, 3, NULL);
  def_builtin("==", (void*)x_eq, 2, NULL);
  def_builtin("!=", (void*)x_neq, 2, NULL);
  def_builtin(">", (void*)x_gt, 2, NULL);
  def_builtin("<", (void*)x_lt, 2, NULL);
  def_builtin("not", (void*)x_not, 1, NULL);
  def_builtin("and", (void*)x_and, 2, NULL);
  def_builtin("all", (void*)x_all, 1, NULL);
  def_builtin("any", (void*)x_any_, 1, NULL);
  def_builtin("or", (void*)x_or, 2, NULL);
  def_builtin("fill", (void*)x_fill, 2, NULL);
  def_builtin("time", (void*)x_time, 0, NULL);
  def_builtin("gc", (void*)x_gc, 0, NULL);
  def_builtin("set", (void*)x_set, 2, NULL);
}

int main(int argc, const char* argv[]) {
  x_any expr;
  x_any value;
  FILE *fp;
  x_env.result = hipStreamCreate(&x_env.stream);

  init();
  if (argc > 1) {
    for (int i = 1; i < argc; i++) {
      if (argv[i][0] == '-') {
        continue;
      }
      else {
        fp = fopen(argv[i], "r");
        if (fp == NULL)
          assert(0);
        for (;;) {
          expr = read_sexpr(fp);
          if (expr == x_env.eof)
            break;
          value = x_eval(expr);
          x_gc();
        }
      }
    }
  }
  else {
    for (;;) {
      printf("? ");
      expr = read_sexpr(stdin);
      if (expr == x_env.eof)
        break;
      value = x_eval(expr);
      printf(": ");
      print_cell(value, stdout);
      putchar('\n');
      x_gc();
    }
  }
  x_env.result = hipStreamDestroy(x_env.stream);
  hipDeviceReset();
}
