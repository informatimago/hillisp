#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
#include <inttypes.h>
#include "lisp.h"

hipStream_t stream;
hipError_t result;

//#define DEBUG 1

#ifdef DEBUG
int debugLevel = 0;
#endif

x_any x_symbol;
x_any x_garbage;
x_any x_nil;
x_any x_true;
x_any x_dot;
x_any x_lparen;
x_any x_rparen;
x_any x_lbrack;
x_any x_rbrack;
x_any x_eof;
x_any x_builtin;
x_any x_token;
x_any x_user;
x_any x_pair;
x_any x_xector;
x_any x_int;
x_any x_fn0;
x_any x_fn1;
x_any x_fn2;
x_any x_fn3;
hash_table_type hash_table;

__device__ __host__ void* bi_malloc(size_t size) {
void* result;
#ifdef __CUDA_ARCH__
  result = malloc(size);
#else
  hipMallocManaged(&result, size);
  hipStreamAttachMemAsync(stream, result);
  SYNCS(stream);
  CHECK;
#endif
  assert(result != NULL);
  return result;
}

char* new_name(const char* name) {
  char *n;
  n = (char*)malloc(strlen(name) + 1);
  strcpy(n, name);
  return n;
}

x_any new_cell(const char* name, x_any type) {
  x_any cell;
  cell = (x_any)malloc(sizeof(x_cell));
  set_cdr(cell, NULL);
  set_car(cell, NULL);
  type(cell) = type;
  if (name == NULL)
    name(cell) = NULL;
  else
    name(cell) = new_name(name);
  return cell;
}

x_any new_xector(const char* name, size_t size) {
  x_any cell;
  x_any_x xector;
  cell = new_cell(name, x_xector);
  xector = (x_any_x)malloc(sizeof(x_xector_t));
  xector->cars = (void**)bi_malloc(size * sizeof(void*));
  xector->size = size;
  set_cdr(cell, xector);
  return cell;
}

int hash(const char *name) {
  int value = 0;
  while (*name != '\0')
    value = (value * X_HASH_MULTIPLIER + *name++) % X_HASH_TABLE_SIZE;
  return value;
}

x_any lookup(const char *name, x_any cell) {
  if (cell == x_nil)
    return NULL;
  else if (strcmp(name(car<x_any>(cell)), name) == 0)
    return car<x_any>(cell);
  else
    return lookup(name, cdr<x_any>(cell));
}

x_any create_symbol(const char *new_name) {
  x_any cell;
  cell = new_cell(new_name, x_symbol);
  if (isdigit(new_name[0]) || (new_name[0] == '-' && isdigit(new_name[1]))) {
    set_car(cell, atol(new_name));
    type(cell) = x_int;
  }
  return cell;
}

void enter(x_any cell) {
  int hash_val;
  hash_val = hash(name(cell));
  hash_table[hash_val] = x_cons(cell, hash_table[hash_val]);
}

x_any intern(const char *name) {
  x_any cell;
  cell = lookup(name, hash_table[hash(name)]);
  if (cell != NULL)
    return cell;
  else {
    cell = create_symbol(name);
    enter(cell);
    return cell;
  }
}

int length(x_any cell) {
  if (cell == x_nil)
    return 0;
  else
    return 1 + length(cdr<x_any>(cell));
}

x_any list_eval(x_any cell) {
  if (cell == x_nil)
    return x_nil;
  if (is_atom(cell))
    return cell;
  else
    return x_cons(x_eval(car<x_any>(cell)), list_eval(cdr<x_any>(cell)));
}

x_any read_token(FILE *infile) {
  int c;
  static char buf[X_MAX_NAME_LEN];
  char *ptr = buf;

  do {
    c = getc(infile);
    if (c == ';')
      do c = getc(infile); while (c != '\n' && c != EOF);
  } while (isspace(c));
  switch (c) {
  case EOF:
    return x_eof;
  case '(':
    return x_lparen;
  case ')':
    return x_rparen;
  case '[':
    return x_lbrack;
  case ']':
    return x_rbrack;
  case '.':
    return x_dot;
  default:
    *ptr++ = c;
    while ((c = getc(infile)) != EOF &&
           !isspace(c) &&
           c != '(' && c != ')' &&
           c != '[' && c!= ']')
      *ptr++ = c;
    if (c != EOF)
      ungetc(c, infile);
    *ptr = '\0';
    if (strcmp(buf, "symbol") == 0)
      return x_symbol;
    return intern(buf);
  }
}

x_any read_cdr(FILE *infile) {
  x_any cdr;
  x_any token;
  cdr = read_sexpr(infile);
  token = read_token(infile);
  if (token == x_rparen)
    return cdr;
  else
    assert(0);
  return x_nil;
}

x_any read_sexpr_tail(FILE *infile) {
  x_any token;
  x_any temp;
  token = read_token(infile);
  if (is_symbol(token) || is_builtin(token))
    return x_cons(token, read_sexpr_tail(infile));
  if (token == x_lparen) {
    temp = read_sexpr_head(infile);
    return x_cons(temp, read_sexpr_tail(infile));
  }
  if (token == x_lbrack) {
    temp = read_xector(infile);
    return x_cons(temp, read_sexpr_tail(infile));
  }
  if (token == x_dot)
    return read_cdr(infile);
  if (token == x_rparen)
    return x_nil;
  if (token == x_eof)
    assert(0);
  return x_nil;
}

x_any read_sexpr_head(FILE *infile) {
  x_any token;
  x_any temp;
  token = read_token(infile);
  if (is_symbol(token) || is_builtin(token))
    return x_cons(token, read_sexpr_tail(infile));
  else if (token == x_lparen) {
    temp = read_sexpr_head(infile);
    return x_cons(temp, read_sexpr_tail(infile));
  }
  else if (token == x_lbrack) {
    temp = read_xector(infile);
    return x_cons(temp, read_sexpr_tail(infile));
  }
  else if (token == x_rparen)
    return x_nil;
  else if (token == x_dot)
    assert(0);
  else if (token == x_eof)
    assert(0);
  return x_nil;
}

x_any read_xector(FILE *infile) {
  x_any val;
  x_any cell;
  x_any typ = NULL;
  size_t size = 0;
  cell = new_xector("xector", X_XECTOR_BLOCK_SIZE);
  do {
    val = x_eval(read_sexpr(infile));
    if (val == x_nil)
      break;
    if (typ == NULL)
      typ = type(val);
    else if (type(val) != typ)
      assert(0); // must all be same type

    if (typ == x_int)
      xector_set_car_ith(cell, size, car<int64_t>(val));
    else if (typ == x_xector)
      xector_set_car_ith(cell, size, car<x_any>(val));
    else
      assert(0);
    size++;
  } while (1);
  xector_size(cell) = size;
  return cell;
}

x_any read_sexpr(FILE *infile) {
  x_any token;
  token = read_token(infile);
  if (is_symbol(token) || is_builtin(token))
    return token;
  if (token == x_lbrack)
    return read_xector(infile);
  if (token == x_lparen)
    return read_sexpr_head(infile);
  if (token == x_rparen)
    assert(0);
  if (token == x_dot)
    assert(0);
  if (token == x_eof)
    return token;
  return x_nil;
}

x_any def_token(const char* new_name) {
  return new_cell(new_name, x_token);
}

x_any def_builtin(char const *name, void *fn, size_t num_args, void *dfn) {
  x_any cell;
  cell = intern(name);
  type(cell) = x_builtin;
  set_cdr(cell, fn);
  set_car(cell, dfn);
  switch(num_args) {
  case 0:
    type(cell) = x_fn0;
    break;
  case 1:
    type(cell) = x_fn1;
    break;
  case 2:
    type(cell) = x_fn2;
    break;
  case 3:
    type(cell) = x_fn3;
    break;
  }
  return cell;
}

void init(void) {
  x_symbol = new_cell("symbol", NULL);
  type(x_symbol) = x_symbol;
  x_pair = new_cell("pair", NULL);
  enter(x_symbol);
  enter(x_pair);

  x_nil = create_symbol("nil");
  for (int i = 0; i < X_HASH_TABLE_SIZE; i++)
    hash_table[i] = x_nil;
  enter(x_nil);

  x_garbage = intern("garbage");
  x_token = intern("token");
  x_builtin = intern("builtin");
  x_user = intern("user");
  x_true = intern("true");
  x_xector = intern("xector");
  x_int = intern("int");
  x_fn0 = intern("fn0");
  x_fn1 = intern("fn1");
  x_fn2 = intern("fn2");
  x_fn3 = intern("fn3");

  x_dot = def_token(".");
  x_lparen = def_token("(");
  x_rparen = def_token(")");
  x_lbrack = def_token("[");
  x_rbrack = def_token("]");
  x_eof = def_token("EOF");

  def_builtin("is", (void*)x_is, 2, NULL);
  def_builtin("isinstance", (void*)x_isinstance, 2, NULL);
  def_builtin("type", (void*)x_type, 1, NULL);
  def_builtin("car", (void*)x_car, 1, NULL);
  def_builtin("cdr", (void*)x_cdr, 1, NULL);
  def_builtin("cons", (void*)x_cons, 2, NULL);
  def_builtin("quote", (void*)x_quote, 1, NULL);
  def_builtin("if", (void*)x_if, 1, NULL);
  def_builtin("while", (void*)x_while, 1, NULL);
  def_builtin("eval", (void*)x_eval, 1, NULL);
  def_builtin("apply", (void*)x_apply, 2, NULL);
  def_builtin("assert", (void*)x_assert, 1, NULL);
  def_builtin("print", (void*)x_print, 1, NULL);
  def_builtin("println", (void*)x_println, 1, NULL);
  def_builtin("+", (void*)x_add, 2, NULL);
  def_builtin("-", (void*)x_sub, 2, NULL);
  def_builtin("*", (void*)x_mul, 2, NULL);
  def_builtin("/", (void*)x_div, 2, NULL);
  def_builtin("fma", (void*)x_fma, 3, NULL);
  def_builtin("==", (void*)x_eq, 2, NULL);
  def_builtin("!=", (void*)x_neq, 2, NULL);
  def_builtin(">", (void*)x_gt, 2, NULL);
  def_builtin("<", (void*)x_lt, 2, NULL);
  def_builtin("not", (void*)x_not, 1, NULL);
  def_builtin("and", (void*)x_and, 2, NULL);
  def_builtin("all", (void*)x_all, 1, NULL);
  def_builtin("any", (void*)x_any_, 1, NULL);
  def_builtin("or", (void*)x_or, 2, NULL);
  def_builtin("fill", (void*)x_fill, 2, NULL);
  def_builtin("time", (void*)x_time, 0, NULL);
}

int main(int argc, const char* argv[]) {
  x_any expr;
  x_any value;
  FILE *fp;
  result = hipStreamCreate(&stream);

  init();
  if (argc > 1) {
    for (int i = 1; i < argc; i++) {
      if (argv[i][0] == '-') {
        continue;
      }
      else {
        fp = fopen(argv[i], "r");
        if (fp == NULL)
          assert(0);
        for (;;) {
          expr = read_sexpr(fp);
          if (expr == x_eof)
            break;
          value = x_eval(expr);
        }
      }
    }
  }
  else {
    for (;;) {
      printf("? ");
      expr = read_sexpr(stdin);
      if (expr == x_eof)
        break;
      value = x_eval(expr);
      printf(": ");
      print_cell(value, stdout);
      putchar('\n');
    }
  }
  result = hipStreamDestroy(stream);
  hipDeviceReset();
}
