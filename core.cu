#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "lisp.h"

template<typename T>
__global__ void
 xd_fill(T* __restrict__ cars, const T val, const size_t size) {
  int i = TID;
  while (i < size) {
    cars[i] = val;
    i += STRIDE;
  }
}

template<typename T>
__global__ void
 xd_all(const T* __restrict__ cell, int* __restrict__ result, const size_t size) {
  if (*result == size)
    if (TID < size)
      if (!cell[TID])
        atomicSub(result, 1);
  __syncthreads();
}

template<typename T>
__global__ void
 xd_any(const T* __restrict__ cell, int* __restrict__ result, const size_t size) {
  if (*result == 0)
    if (TID < size)
      if (cell[TID])
        atomicAdd(result, 1);
  __syncthreads();
}

x_any x_is(x_any cell1, x_any cell2) {
  if (cell1 == cell2)
    return x_env.true_;
  return x_env.nil;
}

x_any x_isinstance(x_any cell1, x_any cell2) {
  do {
    cell1 = type(cell1);
    if (cell1 == cell2)
      return x_env.true_;
  } while(cell1 != x_env.symbol);
  return x_env.nil;
}

x_any x_type(x_any cell) {
  return cell->type;
}

x_any x_assert(x_any cell) {
  assert(cell != x_env.nil);
  return cell;
}

x_any x_car(x_any cell) {
  return car(cell);
}

x_any x_cdr(x_any cell) {
  return cdr(cell);
}

x_any inline x_cons(x_any cell1, x_any cell2) {
  x_any cell;
  cell = new_cell(NULL, x_env.pair);
  set_car(cell, cell1);
  set_cdr(cell, cell2);
  return cell;
}

x_any x_list(x_any args) {
  return args;
}

x_any x_apply(x_any cell, x_any args) {
  x_any expr, result;
  if (is_special(cell))
    return ((x_fn1)val(cell))(args);

  args = eval_list(args);
  if (is_builtin(cell)) {
    if (is_fn1(cell))
      return ((x_fn1)val(cell))(car(args));
    else if (is_fn2(cell))
      return ((x_fn2)val(cell))(car(args), cadr(args));
    else if (is_fn3(cell))
      return ((x_fn3)val(cell))(car(args), cadr(args), caddr(args));
    else if (is_fn0(cell))
      return ((x_fn0)val(cell))();
    else if (is_fnv(cell))
      return ((x_fnv)val(cell))(args);
    else
      assert(0);
  }
  else if (is_user(cell)) {
    expr = car(cell);
    assert(length(args) == length(expr));
    push_frame();

    do {
      local(sval(car(expr)), car(args));
      expr = cdr(expr);
      args = cdr(args);
    } while(expr != x_env.nil);

    expr = cdr(cell);
    do {
      result = x_eval(car(expr));
      expr = cdr(expr);
    } while (expr != x_env.nil);
    pop_frame();
    return result;
  }
  else if (is_symbol(cell) || is_int(cell))
    return x_cons(cell, args);
  else if (is_pair(cell))
    return x_cons(x_eval(cell), args);
  else
    assert(0);
  return x_env.nil;
}

x_any x_quote(x_any args) {
  return car(args);
}

x_any eval_symbol(x_any sym) {
  char* name;
  x_any cell;
  name = sval(sym);
  assert(name != NULL);
  if (isdigit(name[0]) || (name[0] == '-' && isdigit(name[1]))) {
    if (strchr(name, '.') == NULL)
      return new_int(strtoll(name, NULL, 0));
    else
      return new_double(strtod(name, NULL));
  }
  cell = lookup(name, -1);
  if (cell == NULL)
    return sym;
  return car(cell);
}

x_any eval_list(x_any cell) {
  if (cell == x_env.nil)
    return x_env.nil;
  if (is_symbol(cell))
    return eval_symbol(cell);
  else if (is_atom(cell))
    return cell;
  else
    return x_cons(x_eval(car(cell)), eval_list(cdr(cell)));
}

x_any x_eval(x_any cell) {
  x_any temp;
  if (is_symbol(cell))
      return eval_symbol(cell);
  else if (is_atom(cell))
    return cell;
  else if (is_pair(cell)) {
    temp = x_eval(car(cell));
    if (is_func(temp))
      return x_apply(temp, cdr(cell));
    else
      return x_cons(temp, eval_list(cdr(cell)));
  }
  assert(0);
  return x_env.nil;
}

x_any x_not(x_any cell) {
  if (cell == x_env.nil)
    return x_env.true_;
  return x_env.nil;
}

x_any x_and(x_any cell1, x_any cell2) {
  if (cell1 != x_env.nil && cell2 != x_env.nil)
    return x_env.true_;
  return x_env.nil;
}

x_any x_or(x_any cell1, x_any cell2) {
  if (cell1 != x_env.nil || cell2 != x_env.nil)
    return x_env.true_;
  return x_env.nil;
}

x_any x_fill(x_any value, x_any size) {
  x_any cell;
  if (!is_int(size))
    assert(0);
  if (is_int(value)) {
    cell = new_ixector(ival(size));
    xd_fill<int64_t><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<int64_t>(cell), ival(value), xector_size(cell));
  }
  else if (is_double(value)) {
    cell = new_dxector(ival(size));
    xd_fill<double><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
      (cars<double>(cell), fval(value), xector_size(cell));
  }
  CHECK;
  return cell;
}

x_any x_all(x_any cell) {
  int* result;
  if (!is_xector(cell))
    assert(0);
  SYNCS(x_env.stream);
  hipMallocManaged(&result, sizeof(int));
  assert(result != NULL);
  *result = xector_size(cell);
  xd_all<int64_t><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
    (cars<int64_t>(cell), result, xector_size(cell));
  SYNCS(x_env.stream);
  CHECK;
  if (*result != xector_size(cell))
    return x_env.nil;
  return x_env.true_;
}

x_any x_any_(x_any cell) {
  int* result;
  if (!is_xector(cell))
    assert(0);
  SYNCS(x_env.stream);
  hipMallocManaged(&result, sizeof(int));
  assert(result != NULL);
  *result = 0;
  xd_any<int64_t><<<BLOCKS, THREADSPERBLOCK, 0, x_env.stream>>>
    (cars<int64_t>(cell), result, xector_size(cell));
  SYNCS(x_env.stream);
  CHECK;
  if (*result > 0)
    return x_env.true_;
  return x_env.nil;
}

x_any x_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return new_int((tv.tv_sec * 1000) + tv.tv_usec);
}

x_any x_set(x_any args) {
  return bind(sval(car(args)), x_eval(cadr(args)));
}

int64_t inline length(x_any cell) {
  int64_t length = 0;
  if (is_xector(cell))
    return xector_size(cell);
  else if (cdr(cell) == NULL)
    return 0;
  else
    do {
      length += 1;
      cell = cdr(cell);
    } while(cell != x_env.nil);
  return length;
}

x_any x_len(x_any cell) {
  return new_int(length(cell));
}
